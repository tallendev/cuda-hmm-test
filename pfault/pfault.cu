
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "nvml.h"

// Device error checking function
#define devErrChk(ans) { devAssert((ans), __FILE__, __LINE__); }
inline void devAssert(hipError_t code, const char *file, int line, bool abort=true){
	if(code != hipSuccess){
		fprintf(stderr, "Device assert: %s in file \"%s\" line %d\n", hipGetErrorString(code), file, line);
		if(abort) exit(code);
	}
}

// nvml error checking function
#define nvmlErrChk(ans) { nvmlAssert((ans), __FILE__, __LINE__); }
inline void nvmlAssert(nvmlReturn_t code, const char *file, int line, bool abort=true){
	if(code != NVML_SUCCESS){
		fprintf(stderr, "nvml assert: %s in file \"%s\" line %d\n", nvmlErrorString(code), file, line);
		if(abort) exit(code);
	}
}

__global__ void
compute_this(int *pDataFromCpu)
{
    printf ("start\n");
    *pDataFromCpu += 7;
    printf ("yay\n");
}

int main(void)
{
    int devID = 0;
	hipDeviceProp_t devProp;
	devErrChk( hipSetDevice(devID) );
	devErrChk( hipGetDeviceProperties(&devProp, devID) );
	printf("Device name: %s  ID: %d\n", devProp.name, devID);

    //int *pData = (int*)malloc(sizeof(int));
    //if (pData == NULL) { printf("Malloc failed!\n"); exit(1); }
    int *pData = NULL;
    hipMallocManaged(&pData, sizeof(int));
    *pData = 1;

    compute_this<<<1,1>>>(pData);
    if (hipDeviceSynchronize() != hipSuccess)
        printf("Error \n");

    printf("Results: %d\n", *pData);
    hipFree(pData);
    return 0;
}
